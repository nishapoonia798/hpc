
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
using namespace std;

__global__ void multiply(int* A, int* B, int* C, int size) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < size && col < size) {
		int sum = 0;
		for (int i = 0; i < size; i++) {
			sum += A[row * size + i] * B[i * size + col];
		}
		C[row * size + col] = sum;
	}
}

void initialize(int* matrix, int size) {
	for (int i = 0; i < size * size; i++) {
		matrix[i] = rand() % 10;
	}
}

void print(int* matrix, int size) {
	for (int row = 0; row < size; row++) {
		for (int col = 0; col < size; col++) {
			cout << matrix[row * size + col] << " ";
		}
		cout << '\n';
	}
	cout << '\n';
}

int main() {
	int* A, * B, * C;

	int N;
	cout << "Enter matrix size (min. 2): ";
	cin >> N;
	cout << endl;

	int matrixSize = N * N;
	size_t matrixBytes = matrixSize * sizeof(int);

	A = new int[matrixSize];
	B = new int[matrixSize];
	C = new int[matrixSize];

	initialize(A, N);
	initialize(B, N);

	cout << "Matrix A: \n";
	print(A, N);

	cout << "Matrix B: \n";
	print(B, N);

	auto t_start = chrono::high_resolution_clock::now();
		
		int* X, * Y, * Z;
		hipMalloc(&X, matrixBytes);
		hipMalloc(&Y, matrixBytes);
		hipMalloc(&Z, matrixBytes);

		hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);
		hipMemcpy(Y, B, matrixBytes, hipMemcpyHostToDevice);

		int THREADS = 2;

		int BLOCKS = N / THREADS;

		dim3 threads(THREADS, THREADS);
		dim3 blocks(BLOCKS, BLOCKS);

		multiply<<<blocks, threads>>>(X, Y, Z, N);

		hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);


	auto t_end = chrono::high_resolution_clock::now();

	cout << "Multiplication of matrix A and B: \n";
	print(C, N);

	cout
		<< "Time taken: "
		<< chrono::duration_cast<chrono::milliseconds>(t_end - t_start).count()
		<< " ms"
	;

	delete[] A;
	delete[] B;
	delete[] C;

	hipFree(X);
	hipFree(Y);
	hipFree(Z);
}